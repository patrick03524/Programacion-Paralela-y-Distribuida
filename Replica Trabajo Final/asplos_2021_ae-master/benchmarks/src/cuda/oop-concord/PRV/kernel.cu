#include "hip/hip_runtime.h"
__global__ void initContext(GraphChiContext *context, int vertices, int edges) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid == 0) {
    context->setNumIterations(0);
    context->setNumVertices(vertices);
    context->setNumEdges(edges);
  }
}

__global__ void initObject(VirtVertex<float, float> **vertex,
                           GraphChiContext *context, int *row, int *col,
                           int *inrow, int *incol) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    int out_start = row[tid];
    int out_end;
    if (tid + 1 < context->getNumVertices()) {
      out_end = row[tid + 1];
    } else {
      out_end = context->getNumEdges();
    }
    int in_start = inrow[tid];
    int in_end;
    if (tid + 1 < context->getNumVertices()) {
      in_end = inrow[tid + 1];
    } else {
      in_end = context->getNumEdges();
    }
    int indegree = in_end - in_start;
    int outdegree = out_end - out_start;
    vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
    for (int i = in_start; i < in_end; i++) {
      vertex[tid]->setInEdgeV(i - in_start, incol[i], 0.0f);
    }
    // for (int i = out_start; i < out_end; i++) {
    //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
    //}
  }
}

__global__ void initOutEdge(VirtVertex<float, float> **vertex,
                            GraphChiContext *context, int *row, int *col) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    int out_start = row[tid];
    int out_end;
    if (tid + 1 < context->getNumVertices()) {
      out_end = row[tid + 1];
    } else {
      out_end = context->getNumEdges();
    }
    // int in_start = inrow[tid];
    // int in_end;
    // if (tid + 1 < context->getNumVertices()) {
    //    in_end = inrow[tid + 1];
    //} else {
    //    in_end = context->getNumEdges();
    //}
    // int indegree = in_end - in_start;
    // int outdegree = out_end - out_start;
    // vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
    // for (int i = in_start; i < in_end; i++) {
    //    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
    //}
    for (int i = out_start; i < out_end; i++) {
      vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i], 0.0f);
    }
  }
}

__global__ void PageRank(VirtVertex<float, float> **vertex,
                         GraphChiContext *context, int iteration) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    if (iteration == 0) {

      switch (vertex[tid]->type) {
      case 0:
        vertex[tid]->setValueC(1.0f);
        break;
      case 1:
        vertex[tid]->setValueV(1.0f);
        break;
      }
    } else {
      float sum = 0.0f;
      int numInEdge;
	  
	  switch (vertex[tid]->type) {
		case 0:
		numInEdge = vertex[tid]->numInEdgesC();
		  break;
		case 1:
		numInEdge = vertex[tid]->numInEdgesV();
		  break;
		}
      for (int i = 0; i < numInEdge; i++) {
        ChiEdge<float> *inEdge;
		
		switch (vertex[tid]->type) {
			case 0:
			inEdge = vertex[tid]->getInEdgeC(i);
			  break;
			case 1:
			inEdge = vertex[tid]->getInEdgeV(i);
			  break;
			}
		
		
        switch (inEdge->type) {
			case 0:
			sum += inEdge->getValueC();
			  break;
			case 1:
			sum += inEdge->getValueV();
			  break;
			}
      }
      
	  switch (vertex[tid]->type) {
		case 0:
		vertex[tid]->setValueC(0.15f + 0.85f * sum);
		  break;
		case 1:
		vertex[tid]->setValueV(0.15f + 0.85f * sum);
		  break;
		}
      /* Write my value (divided by my out-degree) to my out-edges so neighbors
       * can read it. */
      int numOutEdge;
	 
	  switch (vertex[tid]->type) {
		case 0:
		numOutEdge = vertex[tid]->numOutEdgesC();
		  break;
		case 1:
		numOutEdge = vertex[tid]->numOutEdgesV();
				  break;
		}
	  float outValue;
	  
	  switch (vertex[tid]->type) {
		case 0:
		outValue= vertex[tid]->getValueC() / numOutEdge;
		  break;
		case 1:
		outValue= vertex[tid]->getValueV() / numOutEdge;
				  break;
		}
      for (int i = 0; i < numOutEdge; i++) {
        ChiEdge<float> *outEdge;
		
		switch (vertex[tid]->type) {
			case 0:
			outEdge = vertex[tid]->getOutEdgeC(i);
			  break;
			case 1:
			outEdge = vertex[tid]->getOutEdgeV(i);
					  break;
			}
		
		switch (outEdge->type) {
			case 0:
			outEdge->setValueC(outValue);
			  break;
			case 1:
			outEdge->setValueV(outValue);
			  break;
			}
      }
    }
  }
}

__global__ void copyBack(VirtVertex<float, float> **vertex,
                         GraphChiContext *context, float *pagerank) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    pagerank[tid] = vertex[tid]->getValueV();
  }
}
