#include "hip/hip_runtime.h"
#include "parse_oo.h"

void initContext(GraphChiContext *context, int vertices, int edges) {
    // int tid = blockDim.x * blockIdx.x + threadIdx.x;

    context->setNumIterations(0);
    context->setNumVertices(vertices);
    context->setNumEdges(edges);
}



void part0_initObject(ChiVertex<int, int> **vertex, GraphChiContext *context,
                      int *row, int *col, int *inrow, int *incol,
                      obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        vertex[tid] =
            (ChiVertex<int, int> *)alloc->calloc<ChiVertex<int, int>>(1);
    }
}
void part1_initObject(ChiVertex<int, int> **vertex, GraphChiContext *context,
                      int *row, int *col, int *inrow, int *incol,
                      obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   in_end = inrow[tid + 1];
        // } else {
        //   in_end = context->getNumEdges();
        // }
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid].inEdgeDataArray =
        //     (ChiEdge<myType> *)alloc->my_new<Edge<myType>>(indegree);
        // vertex[tid].outEdgeDataArray =
        //     (ChiEdge<myType> **)alloc->my_new<Edge<myType> *>(outdegree);
        // new (&vertex[tid]) ChiVertex<int, int>(tid, indegree,
        // outdegree,alloc);
        vertex[tid]->set_in_out(alloc);
        // vertex[tid].setValue(INT_MAX);
        // for (int i = in_start; i < in_end; i++) {
        //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
        // }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}
__global__ void part_kern0_initObject(ChiVertex<int, int> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;

        new (vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree);

        // vertex[tid].setValue(INT_MAX);
        // for (int i = in_start; i < in_end; i++) {
        //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
        // }
    }
}
__global__ void part_kern1_initObject(ChiVertex<int, int> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }

        vertex[tid]->setValue(INT_MAX);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid]->setInEdge(i - in_start, incol[i], INT_MAX);
        }
    }
}
void initOutEdge(ChiVertex<int, int> **vertex, GraphChiContext *context,
                 int *row, int *col) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i],
                                    INT_MAX);
        }
    }
}


__global__ void kern_initOutEdge(ChiVertex<int, int> **vertex,
                                 GraphChiContext *context, int *row, int *col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //    in_end = inrow[tid + 1];
        //} else {
        //    in_end = context->getNumEdges();
        //}
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
        // for (int i = in_start; i < in_end; i++) {
        //    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
        //}

        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i],
                                    INT_MAX);
        }
    }
}

__managed__ __align__(16) char buf2[128];
template <class myType>
__global__ void vptrPatch(myType *array, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("-----\n");
    myType *obj;
    obj = new (buf2) myType();
    // void *p;
    // p=(void *)0x111111111;
    // memcpy(p, obj, sizeof(void *));
    // printf("---%p--\n", p);
    if (tid < n) {
        memcpy(&array[tid], obj, sizeof(void *));
        // printf("---%p--\n",p);
    }
}

__global__ void vptrPatch_Edge(ChiVertex<int, int> *vertex, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    Edge<int> *obj;
    obj = new (buf2) Edge<int>();

    if (tid < n)
        if (tid == 0)
            vertex[tid].vptrPatch(obj, 1);
        else
            vertex[tid].vptrPatch(obj, 1);
}

__managed__ obj_info_tuble *vfun_table;
__managed__ unsigned tree_size_g;
__managed__ void *temp_copyBack;
__managed__ void *temp_TP;

__global__ void BFS(ChiVertex<int, int> **vertex, GraphChiContext *context,
                    int iteration) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    void **vtable;
    if (tid < context->getNumVertices()) {
        if (iteration == 0) {
            if (tid == 0) {
                vertex[tid]->setValue(0);
                int numOutEdge;
                numOutEdge = vertex[tid]->numOutEdges();
                for (int i = 0; i < numOutEdge; i++) {
                    ChiEdge<int> *outEdge;
                    outEdge = vertex[tid]->getOutEdge(i);
                    vtable = get_vfunc_type(outEdge,vfun_table);
                    temp_TP = vtable[2];
                    CLEANPTR(outEdge, ChiEdge<int> *)->setValue(1);
                    //outEdge->setValue(1);
                }
            }
        } else {
            int curmin;
            curmin = vertex[tid]->getValue();
            int numInEdge;
            numInEdge = vertex[tid]->numInEdges();
            for (int i = 0; i < numInEdge; i++) {
                ChiEdge<int> *inEdge;
                inEdge = vertex[tid]->getInEdge(i);
                vtable = get_vfunc_type(inEdge,vfun_table);
                temp_TP = vtable[1];
                 curmin = min(curmin,  CLEANPTR(inEdge,ChiEdge<int> *)->getValue());
                //curmin = min(curmin,  inEdge->getValue());
            }
            int vertValue;
            vertValue = vertex[tid]->getValue();
            if (curmin < vertValue) {
                vertex[tid]->setValue(curmin);
                int numOutEdge;
                numOutEdge = vertex[tid]->numOutEdges();
                for (int i = 0; i < numOutEdge; i++) {
                    ChiEdge<int> *outEdge;
                    outEdge = vertex[tid]->getOutEdge(i);
                    int edgeValue;
                    vtable = get_vfunc_type(outEdge,vfun_table);
                    temp_TP = vtable[1];
                     edgeValue =  CLEANPTR(outEdge, ChiEdge<int> *)->getValue();
                    //edgeValue =  outEdge->getValue();
                    if (edgeValue > curmin + 1) {
                        vtable = get_vfunc_type(outEdge,vfun_table);
                        temp_TP = vtable[2];
                        CLEANPTR(outEdge, ChiEdge<int> *)->setValue(curmin + 1);
                    }
                }
            }
        }
    }
}

__global__ void copyBack(ChiVertex<int, int> **vertex, GraphChiContext *context,
                         int *index) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        index[tid] = vertex[tid]->getValue();
    }
}
