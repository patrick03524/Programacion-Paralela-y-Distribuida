#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <chrono>

#include "../configuration.h"
#include "../dataset_loader.h"
#include "generation.h"
//#include "../rendering.h"

#define gpuErrchk(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort) exit(code);
    }
}
#ifdef OPTION_RENDER
// Rendering array.
// TODO: Fix variable names.
__device__ int *device_render_cells;
int *host_render_cells;
int *d_device_render_cells;
#endif  // OPTION_RENDER

// Dataset.
__device__ int SIZE_X;
__device__ int SIZE_Y;
__managed__ CellV **cells;
__managed__ Cell *cells2;
dataset_t dataset;

// Only count alive agents in state 0.
__device__ int num_alive_neighbors(AgentV *ptr) {
    int cell_x = ptr->cell_id_ % SIZE_X;
    int cell_y = ptr->cell_id_ / SIZE_X;
    int result = 0;

    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = cell_x + dx;
            int ny = cell_y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                AgentV *ptr = cells[ny * SIZE_X + nx]->agent();
                AgentV *alive = nullptr;
                if (ptr)
                    if (ptr->isAlive())

                        alive = cells[ny * SIZE_X + nx]->agent();

                if (alive != nullptr && alive->is_state_equal(0)) {
                    result++;
                }
            }
        }
    }

    return result;
}

__device__ void create_candidates(AgentV *ptr) {
    assert(ptr->is_new());

    // TODO: Consolidate with Agent::num_alive_neighbors().
    int cell_x = ptr->cell_id_ % SIZE_X;
    int cell_y = ptr->cell_id_ / SIZE_X;

    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = cell_x + dx;
            int ny = cell_y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                auto cid = ny * SIZE_X + nx;
                if (cells[cid]->is_empty()) {
                    if (atomicCAS(&cells[cid]->reserved, 0, 1) == 0) {
                        cells[cid]->set_agent(cid, AgentType::isCandidate);
                    }
                }
            }
        }
    }
}

__global__ void create_cells() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        cells[i] = new (&cells2[i]) Cell();
        assert(cells[i] != nullptr);
    }
}

// Must be followed by Agent::update().
__global__ void load_game(int *cell_ids, int num_cells) {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < num_cells;
         i += blockDim.x * gridDim.x) {
        cells[cell_ids[i]]->set_agent(cell_ids[i], AgentType::isAlive);
        assert(cells[cell_ids[i]]->agent() != nullptr);
        assert(cells[cell_ids[i]]->agent()->cell_id() == cell_ids[i]);
    }
}

__device__ void alive_update_2(int i) {
    AgentV *ptr = cells[i]->agent();
    if (ptr)
        if (ptr->isAlive()) {
            int cid = ptr->cell_id_;

            // TODO: Consider splitting in two classes for less divergence.
            if (ptr->is_new()) {
                // Create candidates in neighborhood.
                create_candidates(ptr);
            } else {
                if (ptr->get_action() == kActionDie && ptr->is_state_equal(0)) {
                    // Increment state. If reached max. state, replace with
                    // Candidate.
                    ptr->inc_state();
                    ptr->set_action(kActionNone);
                } else if (ptr->is_state_in_range(0, kNumStates)) {
                    ptr->inc_state();
                } else if (ptr->is_state_equal(kNumStates)) {
                    // Replace with Candidate.
                    cells[cid]->set_agent(cid, AgentType::isCandidate);
                    // delete this;
                }
            }
        }
}

__global__ void update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        if (ptr)
            if (ptr->isCandidate()) {
                int cid = ptr->cell_id_;

                if (ptr->get_action() == kActionSpawnAlive) {
                    cells[cid]->set_agent(cid, AgentType::isAlive);
                    // delete this;
                } else if (ptr->get_action() == kActionDie) {
                    cells[cid]->delete_agent();
                    cells[cid]->reserved = 0;
                    // delete this;
                }
            }
        // alive_update_2(i);
    }
}

__global__ void prepare() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        if (ptr)

            if (ptr->isAlive()) {
                if (ptr->is_state_equal(0)) {
                    ptr->set_is_new(false);

                    // Also counts this object itself.
                    int alive_neighbors = num_alive_neighbors(ptr) - 1;

                    const bool stay_alive_param[9] = kStayAlive;
                    if (!stay_alive_param[alive_neighbors]) {
                        ptr->set_action(kActionDie);
                    }
                }
            }
        ptr = cells[i]->agent();
        if (ptr)
            if (ptr->isCandidate()) {
                int alive_neighbors = num_alive_neighbors(ptr);
                const bool spawn_param[9] = kSpawnNew;

                if (spawn_param[alive_neighbors]) {
                    ptr->set_action(kActionSpawnAlive);
                } else if (alive_neighbors == 0) {
                    ptr->set_action(kActionDie);
                }
            }
    }
}

__global__ void update_checksum() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        if (ptr) ptr->update_checksum();
    }
}

__global__ void alive_update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        if (ptr)
            if (ptr->isAlive()) {
                int cid = ptr->cell_id_;

                // TODO: Consider splitting in two classes for less divergence.
                if (ptr->is_new()) {
                    // Create candidates in neighborhood.
                    create_candidates(ptr);
                } else {
                    bool flag1 = ptr->get_action() == kActionDie;

                    flag1 = flag1 && ptr->is_state_equal(0);

                    bool flag2 = ptr->is_state_in_range(0, kNumStates);

                    bool flag3 = ptr->is_state_equal(kNumStates);
                    if (flag1) {
                        // Increment state. If reached max. state, replace with
                        // Candidate.

                        ptr->inc_state();

                        ptr->set_action(kActionNone);
                    } else if (flag2) {
                        ptr->inc_state();
                    } else if (flag3) {
                        // Replace with Candidate.

                        cells[cid]->set_agent(cid, AgentType::isCandidate);
                        // delete this;
                    }
                }
            }
    }
}

void transfer_dataset() {
    int *dev_cell_ids;
    int num_alive = dataset.alive_cells.size();
    printf("number of alive %d \n", num_alive);
    hipMalloc(&dev_cell_ids, sizeof(int) * num_alive);
    hipMemcpy(dev_cell_ids, dataset.alive_cells.data(),
               sizeof(int) * num_alive, hipMemcpyHostToDevice);

#ifndef NDEBUG
    printf("Loading on GPU: %i alive cells.\n", num_alive);
#endif  // NDEBUG

    load_game<<<1024, 1024>>>(dev_cell_ids, num_alive);
    gpuErrchk(hipDeviceSynchronize());
    // hipFree(dev_cell_ids);

    alive_update<<<1024, 1024>>>();
    gpuErrchk(hipDeviceSynchronize());
}

__device__ int device_checksum;
__device__ int device_num_candidates;

__device__ __noinline__ void Agent::update_checksum() {
    if (this->isAlive())
        atomicAdd(&device_checksum, 1);
    else
        atomicAdd(&device_num_candidates, 1);
}

int checksum() {
    int host_checksum = 0;
    int host_num_candidates = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(device_checksum), &host_checksum, sizeof(int), 0,
                       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(device_num_candidates), &host_num_candidates, sizeof(int),
                       0, hipMemcpyHostToDevice);

    update_checksum<<<1024, 1024>>>();
    gpuErrchk(hipDeviceSynchronize());

    hipMemcpyFromSymbol(&host_checksum, HIP_SYMBOL(device_checksum), sizeof(int), 0,
                         hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&host_num_candidates, HIP_SYMBOL(device_num_candidates),
                         sizeof(int), 0, hipMemcpyDeviceToHost);

    return host_checksum + host_num_candidates;
}

int main(int /*argc*/, char ** argv) {
    // Load data set.
    dataset = load_burst();

    hipMemcpyToSymbol(HIP_SYMBOL(SIZE_X), &dataset.x, sizeof(int), 0,
                       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(SIZE_Y), &dataset.y, sizeof(int), 0,
                       hipMemcpyHostToDevice);

#ifdef OPTION_RENDER
    init_renderer();
#endif  // OPTION_RENDER

    // Allocate memory.

    hipDeviceSetLimit(hipLimitMallocHeapSize, 4ULL * 1024 * 1024 * 1024);
    mem_alloc shared_mem(4ULL * 1024 * 1024 * 1024);
    obj_alloc my_obj_alloc(&shared_mem, atoll(argv[1]));
    // hipMalloc(&cells, sizeof(Cell *) * dataset.x * dataset.y);
    // hipMalloc(&cells2, sizeof(Cell) * dataset.x * dataset.y);
    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    cells = (CellV **)my_obj_alloc.calloc<CellV *>(dataset.x * dataset.y);
    for (int i = 0; i < dataset.x * dataset.y; i++) {
        cells[i] = (Cell *)my_obj_alloc.my_new<Cell>();
        cells[i]->inst_cell(&my_obj_alloc);
        // assert(cells[i] != nullptr);
    }
    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    my_obj_alloc.toDevice();
    high_resolution_clock::time_point t3 = high_resolution_clock::now();
    duration<double> alloc_time = duration_cast<duration<double>>(t2 - t1);
    duration<double> vptr_time = duration_cast<duration<double>>(t3 - t2);
  
    printf("alloc_time : %f \nvptr patching : %f \n",alloc_time.count(),vptr_time.count() );
    printf("number of objs:%d\n", dataset.x * dataset.y);

    // Initialize cells.
    // create_cells<<<1024, 1024>>>();
    gpuErrchk(hipDeviceSynchronize());

    transfer_dataset();

    auto time_start = std::chrono::system_clock::now();

    // Run simulation.
    for (int i = 0; i < kNumIterations; ++i) {
#ifdef OPTION_RENDER
        render();
#endif  // OPTION_RENDER

#ifndef NDEBUG
        if (i % 30 == 0) printf("%i\n", i);

#endif  // NDEBUG

        // can_prepare<<<1024, 1024>>>();
        // gpuErrchk(hipDeviceSynchronize());
        prepare<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());
        update<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());
        alive_update<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());
        // alive_update<<<1024, 1024>>>();
        // gpuErrchk(hipDeviceSynchronize());
    }

    auto time_end = std::chrono::system_clock::now();
    auto elapsed = time_end - time_start;
    auto micros =
        std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

#ifdef OPTION_RENDER
    close_renderer();
#endif  // OPTION_RENDER

#ifndef NDEBUG
    printf("Checksum: %i \n", checksum());
#endif  // NDEBUG

    printf("%lu, \n", micros);

    //  if (kOptionPrintStats) {
    //    allocator_handle->DBG_print_collected_stats();
    //  }

#ifdef OPTION_RENDER
    delete[] host_render_cells;
    hipFree(d_device_render_cells);
#endif  // OPTION_RENDER

    return 0;
}
