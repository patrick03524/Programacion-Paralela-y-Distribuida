#include "hip/hip_runtime.h"
#include <chrono>

#include "../configuration.h"
#include "../dataset_loader.h"

#include "gol.h"

#ifdef OPTION_RENDER
// Rendering array.
// TODO: Fix variable names.
__device__ char *device_render_cells;
char *host_render_cells;
char *d_device_render_cells;
#endif  // OPTION_RENDER

// Dataset.
__device__ int SIZE_X;
__device__ int SIZE_Y;
__device__ Cell **cells;
dataset_t dataset;

__device__ int num_alive_neighbors(AgentV *ptr) {
    // CONCORD
    int cell_x;
    CONCORD(cell_x, ptr, cell_id());
    cell_x = cell_x % SIZE_X;
    // CONCORD
    int cell_y;
    CONCORD(cell_y, ptr, cell_id());
    cell_y = cell_y / SIZE_X;
    int result = 0;

    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = cell_x + dx;
            int ny = cell_y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                // CONCORD
                AgentV *tmp;
                CONCORD(tmp, cells[ny * SIZE_X + nx], agent());
                ;

                if (tmp) {
                    // CONCORD
                    bool cond;
                    CONCORD(cond, tmp, isAlive())
                    if (cond) {
                        result++;
                    }
                }
            }
        }
    }

    return result;
}

__device__ void maybe_create_candidate(AgentV *ptr, int x, int y) {
    // Check neighborhood of cell to determine who should create Candidate.
    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = x + dx;
            int ny = y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                // CONCORD
                AgentV *alive;
                CONCORD(alive, cells[ny * SIZE_X + nx], agent());
                ;
                if (alive != nullptr) {
                    // CONCORD
                    bool cond;
                    CONCORD(cond, alive, is_new())
                    if (cond) {
                        if (alive == ptr) {
                            // Create candidate now.
                            // CONCORD
                            CONCORD(cells[y * SIZE_X + x],
                                    set_agent((y * SIZE_X + x),
                                              AgentType::isCandidate));

                        }  // else: Created by other thread.

                        return;
                    }
                }
            }
        }
    }

    assert(false);
}
__device__ void create_candidates(AgentV *ptr) {
    // CONCORD
    bool cond;
    CONCORD(cond, ptr, is_new())
    //assert(cond);
    // CONCORD
    CONCORD(cond, ptr, isAlive())
    assert(cond);
    // TODO: Consolidate with Agent::num_alive_neighbors().
    // CONCORD
    int cell_x;
    CONCORD(cell_x, ptr, cell_id());
    cell_x = cell_x % SIZE_X;
    // CONCORD
    int cell_y;
    CONCORD(cell_y, ptr, cell_id());
    cell_y = cell_y / SIZE_X;

    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = cell_x + dx;
            int ny = cell_y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                // CONCORD
                bool cond2;
                CONCORD(cond2, cells[ny * SIZE_X + nx], is_empty())
                if (cond2) {
                    // Candidate should be created here.
                    maybe_create_candidate(ptr, nx, ny);
                }
            }
        }
    }
}

__device__ void Alive_prepare(AgentV *ptr) {
    if (ptr) {
        // CONCORD
        bool cond2;
        CONCORD(cond2, ptr, isAlive())
        if (cond2) {
            // CONCORD
            CONCORD(ptr, set_is_new(false));
            ;

            // Also counts this object itself.
            int alive_neighbors = num_alive_neighbors(ptr) - 1;

            if (alive_neighbors < 2 || alive_neighbors > 3) {
                // CONCORD
                CONCORD(ptr, set_action(kActionDie));
                ;
            }
        }
    }
}

__device__ void Alive_update(AgentV *ptr) {
    if (ptr) {
        // CONCORD
        bool cond2;
        CONCORD(cond2, ptr, isAlive())
        if (cond2) {
            // CONCORD
            int cid;
            CONCORD(cid, ptr, cell_id());

            // TODO: Consider splitting in two classes for less divergence.
            // CONCORD
            bool cond3;
            CONCORD(cond3, ptr, isAlive())
            if (cond3) {
                // Create candidates in neighborhood.
                create_candidates(ptr);
            } else {
                // CONCORD
                int act;
                CONCORD(act, ptr, get_action())
                if (act == kActionDie) {
                    // CONCORD
                    CONCORD(cells[cid], set_agent(cid, AgentType::isCandidate));
                }
            }
        }
    }
}

__global__ void alive_prepare() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        // CONCORD
        AgentV *ptr;
        CONCORD(ptr, cells[i], agent());

        Alive_prepare(ptr);
    }
}

__global__ void alive_update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        // CONCORD
        AgentV *ptr;
        CONCORD(ptr, cells[i], agent());

        Alive_update(ptr);
    }
}
__device__ void Candidate_prepare(AgentV *ptr) {
    if (ptr) {
        // CONCORD
        bool cond2;
        CONCORD(cond2, ptr, isCandidate())
        if (cond2) {
            int alive_neighbors = num_alive_neighbors(ptr);

            if (alive_neighbors == 3) {
                // CONCORD
                CONCORD(ptr, set_action(kActionSpawnAlive));
                ;

            } else if (alive_neighbors == 0) {
                // CONCORD
                CONCORD(ptr, set_action(kActionDie));
            }
        }
    }
}

__device__ void Candidate_update(AgentV *ptr) {
    // TODO: Why is this necessary?
    if (ptr) {
        // CONCORD
        bool cond2;
        CONCORD(cond2, ptr, isCandidate())
        if (cond2) {
            // CONCORD
            int cid;
            CONCORD(cid, ptr, cell_id());

            // CONCORD
            int act;
            CONCORD(act, ptr, get_action())
            int act2;
            CONCORD(act2, ptr, get_action())
            if (act == kActionSpawnAlive) {
                // CONCORD
                CONCORD(cells[cid], set_agent(cid, AgentType::isAlive));

                // CONCORD
            } else if (act2 == kActionDie) {
                // CONCORD
                CONCORD(cells[cid], delete_agent());
            }
        }
    }
}

__global__ void candidate_prepare() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        // CONCORD
        AgentV *ptr;
        CONCORD(ptr, cells[i], agent());

        Candidate_prepare(ptr);
    }
}

__global__ void candidate_update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        // CONCORD
        AgentV *ptr;
        CONCORD(ptr, cells[i], agent());

        Candidate_update(ptr);
    }
}
__global__ void create_cells() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        cells[i] = new Cell();
        assert(cells[i] != nullptr);
    }
}

// Must be followed by Alive::update().
__global__ void load_game(int *cell_ids, int num_cells) {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < num_cells;
         i += blockDim.x * gridDim.x) {
        // CONCORD
        CONCORD(cells[cell_ids[i]], set_agent(cell_ids[i], AgentType::isAlive));

        // CONCORD

        assert(cells[cell_ids[i]]->agent()->cell_id() == cell_ids[i]);
    }
}
int checksum();
void transfer_dataset() {
    int *dev_cell_ids;
    hipMalloc(&dev_cell_ids, sizeof(int) * dataset.num_alive);
    hipMemcpy(dev_cell_ids, dataset.alive_cells,
               sizeof(int) * dataset.num_alive, hipMemcpyHostToDevice);

#ifndef NDEBUG
    printf("Loading on GPU: %i alive cells.\n", dataset.num_alive);
#endif  // NDEBUG

    load_game<<<128, 128>>>(dev_cell_ids, dataset.num_alive);
    gpuErrchk(hipDeviceSynchronize());
    hipFree(dev_cell_ids);

    alive_update<<<1024, 1024>>>();
    gpuErrchk(hipDeviceSynchronize());
}

__device__ int device_checksum;
__device__ int device_num_candidates;

__device__ __noinline__ void AgentV::update_checksum() {
    // CONCORD
    if (this->isAlive()) atomicAdd(&device_checksum, 1);
    // CONCORD
    else if (this->isCandidate())
        atomicAdd(&device_num_candidates, 1);
}
__global__ void update_checksum() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        // CONCORD
        AgentV *ptr;
        CONCORD(ptr, cells[i], agent());

        if (ptr) {
            // CONCORD
            CONCORD(ptr, update_checksum());
        }
    }
}
int checksum() {
    int host_checksum = 0;
    int host_num_candidates = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(device_checksum), &host_checksum, sizeof(int), 0,
                       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(device_num_candidates), &host_num_candidates, sizeof(int),
                       0, hipMemcpyHostToDevice);

    // CONCORD
    // allocator_handle->parallel_do<Alive, &Alive::update_checksum>();
    // allocator_handle->parallel_do<Candidate, &Candidate::update_counter>();
    update_checksum<<<1024, 1024>>>();
    hipMemcpyFromSymbol(&host_checksum, HIP_SYMBOL(device_checksum), sizeof(int), 0,
                         hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&host_num_candidates, HIP_SYMBOL(device_num_candidates),
                         sizeof(int), 0, hipMemcpyDeviceToHost);

    return host_checksum;
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s filename.pgm\n", argv[0]);
        exit(1);
    } else {
        // Load data set.
        dataset = load_from_file(argv[1]);
    }
    hipDeviceSetLimit(hipLimitMallocHeapSize, 4ULL * 1024 * 1024 * 1024);

    hipMemcpyToSymbol(HIP_SYMBOL(SIZE_X), &dataset.x, sizeof(int), 0,
                       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(SIZE_Y), &dataset.y, sizeof(int), 0,
                       hipMemcpyHostToDevice);

    // Allocate memory.
    Cell **host_cells;
    hipMalloc(&host_cells, sizeof(Cell *) * dataset.x * dataset.y);
    hipMemcpyToSymbol(HIP_SYMBOL(cells), &host_cells, sizeof(Cell **), 0,
                       hipMemcpyHostToDevice);

    // Initialize cells.
    create_cells<<<128, 128>>>();
    gpuErrchk(hipDeviceSynchronize());

    transfer_dataset();

    auto time_start = std::chrono::system_clock::now();
    printf("Checksum: %i\n", checksum());
    // Run simulation.
    for (int i = 0; i < kNumIterations; ++i) {
        candidate_prepare<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        alive_prepare<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        candidate_update<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        alive_update<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        // printf("Checksum: %i\n", checksum());
    }

    auto time_end = std::chrono::system_clock::now();
    auto elapsed = time_end - time_start;
    auto micros =
        std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

    printf("Checksum: %i\n", checksum());

    printf("%lu, \n", micros);

    hipFree(host_cells);

    return 0;
}
