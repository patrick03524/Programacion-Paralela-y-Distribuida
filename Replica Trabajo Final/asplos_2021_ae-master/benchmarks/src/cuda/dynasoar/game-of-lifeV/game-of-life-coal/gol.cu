#include "hip/hip_runtime.h"
#include <chrono>

#include "../configuration.h"
#include "../dataset_loader.h"

#include "gol.h"

#ifdef OPTION_RENDER
// Rendering array.
// TODO: Fix variable names.
__device__ char *device_render_cells;
char *host_render_cells;
char *d_device_render_cells;
#endif  // OPTION_RENDER

// Dataset.
__device__ int SIZE_X;
__device__ int SIZE_Y;
__managed__ CellV **cells;
dataset_t dataset;

__device__ int num_alive_neighbors(AgentV *ptr) {
    void **vtable;

    COAL_AgentV_cell_id(ptr);
    int cell_x = ptr->cell_id() % SIZE_X;
    COAL_AgentV_cell_id(ptr);
    int cell_y = ptr->cell_id() / SIZE_X;
    int result = 0;

    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = cell_x + dx;
            int ny = cell_y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                COAL_CellV_agent(cells[ny * SIZE_X + nx]);
                AgentV *tmp = cells[ny * SIZE_X + nx]->agent();

                if (tmp) {
                    COAL_AgentV_isAlive(tmp);
                    if (tmp->isAlive()) {
                        result++;
                    }
                }
            }
        }
    }

    return result;
}

__device__ void maybe_create_candidate(AgentV *ptr, int x, int y) {
    void **vtable;
    // Check neighborhood of cell to determine who should create Candidate.
    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = x + dx;
            int ny = y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                COAL_CellV_agent(cells[ny * SIZE_X + nx]);
                AgentV *alive = cells[ny * SIZE_X + nx]->agent();
                if (alive != nullptr) {
                    COAL_AgentV_is_new(alive);
                    if (alive->is_new()) {
                        if (alive == ptr) {
                            // Create candidate now.
                            COAL_CellV_set_agent(cells[y * SIZE_X + x]);
                            cells[y * SIZE_X + x]->set_agent(
                                (y * SIZE_X + x), AgentType::isCandidate);

                        }  // else: Created by other thread.

                        return;
                    }
                }
            }
        }
    }

    assert(false);
}
__device__ void create_candidates(AgentV *ptr) {
    void **vtable;
    COAL_AgentV_is_new(ptr);
    assert(ptr->is_new());
    COAL_AgentV_isAlive(ptr) assert(ptr->isAlive());
    // TODO: Consolidate with Agent::num_alive_neighbors().
    COAL_AgentV_cell_id(ptr);
    int cell_x = ptr->cell_id() % SIZE_X;
    COAL_AgentV_cell_id(ptr);
    int cell_y = ptr->cell_id() / SIZE_X;

    for (int dx = -1; dx < 2; ++dx) {
        for (int dy = -1; dy < 2; ++dy) {
            int nx = cell_x + dx;
            int ny = cell_y + dy;

            if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
                COAL_CellV_is_empty(cells[ny * SIZE_X + nx]);
                if (cells[ny * SIZE_X + nx]->is_empty()) {
                    // Candidate should be created here.
                    maybe_create_candidate(ptr, nx, ny);
                }
            }
        }
    }
}

__device__ void Alive_prepare(AgentV *ptr) {
    void **vtable;

    if (ptr) {
        COAL_AgentV_isAlive(ptr);
        if (ptr->isAlive()) {
            COAL_AgentV_set_is_new(ptr);
            ptr->set_is_new(false);

            // Also counts this object itself.
            int alive_neighbors = num_alive_neighbors(ptr) - 1;

            if (alive_neighbors < 2 || alive_neighbors > 3) {
                COAL_AgentV_set_action(ptr);
                ptr->set_action(kActionDie);
            }
        }
    }
}

__device__ void Alive_update(AgentV *ptr) {
    void **vtable;
    if (ptr) {
        COAL_AgentV_isAlive(ptr);

        if (ptr->isAlive()) {
            COAL_AgentV_cell_id(ptr);

            int cid = ptr->cell_id();

            // TODO: Consider splitting in two classes for less divergence.
            COAL_AgentV_is_new(ptr);
            if (ptr->is_new()) {
                // Create candidates in neighborhood.
                create_candidates(ptr);
            } else {
                COAL_AgentV_get_action(ptr);
                if (ptr->get_action() == kActionDie) {
                    COAL_CellV_set_agent(cells[cid]);
                    cells[cid]->set_agent(cid, AgentType::isCandidate);
                }
            }
        }
    }
}

__global__ void alive_prepare() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        Alive_prepare(ptr);
    }
}

__global__ void alive_update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        Alive_update(ptr);
    }
}
__device__ void Candidate_prepare(AgentV *ptr) {
    void **vtable;
    if (ptr) {
        COAL_AgentV_isCandidate(ptr);
        if (ptr->isCandidate()) {
            int alive_neighbors = num_alive_neighbors(ptr);

            if (alive_neighbors == 3) {
                COAL_AgentV_set_action(ptr);
                ptr->set_action(kActionSpawnAlive);

            } else if (alive_neighbors == 0) {
                COAL_AgentV_set_action(ptr);
                ptr->set_action(kActionDie);
            }
        }
    }
}

__device__ void Candidate_update(AgentV *ptr) {
    // TODO: Why is this necessary?
    void **vtable;

    if (ptr) {
        COAL_AgentV_isCandidate(ptr);

        if (ptr->isCandidate()) {
            COAL_AgentV_cell_id(ptr);

            int cid = ptr->cell_id();
            COAL_AgentV_get_action(ptr);
            if (ptr->get_action() == kActionSpawnAlive) {
              COAL_CellV_set_agent(cells[cid]);
                cells[cid]->set_agent(cid, AgentType::isAlive);
            } else {
              COAL_AgentV_get_action(ptr);
                if (ptr->get_action() == kActionDie) {
                  COAL_CellV_delete_agent(cells[cid]);
                    cells[cid]->delete_agent();
                }
            }
        }
    }
}

__global__ void candidate_prepare() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        Candidate_prepare(ptr);
    }
}

__global__ void candidate_update() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        Candidate_update(ptr);
    }
}
__global__ void create_cells() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        cells[i] = new Cell();
        assert(cells[i] != nullptr);
    }
}

// Must be followed by Alive::update().
__global__ void load_game(int *cell_ids, int num_cells) {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < num_cells;
         i += blockDim.x * gridDim.x) {
        cells[cell_ids[i]]->set_agent(cell_ids[i], AgentType::isAlive);
        assert(cells[cell_ids[i]]->agent()->cell_id() == cell_ids[i]);
    }
}
int checksum();
void transfer_dataset() {
    int *dev_cell_ids;
    hipMalloc(&dev_cell_ids, sizeof(int) * dataset.num_alive);
    hipMemcpy(dev_cell_ids, dataset.alive_cells,
               sizeof(int) * dataset.num_alive, hipMemcpyHostToDevice);

#ifndef NDEBUG
    printf("Loading on GPU: %i alive cells.\n", dataset.num_alive);
#endif  // NDEBUG

    load_game<<<128, 128>>>(dev_cell_ids, dataset.num_alive);
    gpuErrchk(hipDeviceSynchronize());
    hipFree(dev_cell_ids);

    alive_update<<<1024, 1024>>>();
    gpuErrchk(hipDeviceSynchronize());
}

__device__ int device_checksum;
__device__ int device_num_candidates;

__device__ void Agent::update_checksum() {
    if (this->isAlive())
        atomicAdd(&device_checksum, 1);
    else if (this->isCandidate())
        atomicAdd(&device_num_candidates, 1);
}
__global__ void update_checksum() {
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < SIZE_X * SIZE_Y;
         i += blockDim.x * gridDim.x) {
        AgentV *ptr = cells[i]->agent();
        if (ptr) ptr->update_checksum();
    }
}
int checksum() {
    int host_checksum = 0;
    int host_num_candidates = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(device_checksum), &host_checksum, sizeof(int), 0,
                       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(device_num_candidates), &host_num_candidates, sizeof(int),
                       0, hipMemcpyHostToDevice);

    // allocator_handle->parallel_do<Alive, &Alive::update_checksum>();
    // allocator_handle->parallel_do<Candidate, &Candidate::update_counter>();
    update_checksum<<<1024, 1024>>>();
    hipMemcpyFromSymbol(&host_checksum, HIP_SYMBOL(device_checksum), sizeof(int), 0,
                         hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&host_num_candidates, HIP_SYMBOL(device_num_candidates),
                         sizeof(int), 0, hipMemcpyDeviceToHost);

    return host_checksum;
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Usage: %s filename.pgm\n", argv[0]);
        exit(1);
    } else {
        // Load data set.
        dataset = load_from_file(argv[1]);
    }
    hipDeviceSetLimit(hipLimitMallocHeapSize, 4ULL * 1024 * 1024 * 1024);
    mem_alloc shared_mem(4ULL * 1024 * 1024 * 1024);
    obj_alloc my_obj_alloc(&shared_mem, atoll(argv[2]));
    hipMemcpyToSymbol(HIP_SYMBOL(SIZE_X), &dataset.x, sizeof(int), 0,
                       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(SIZE_Y), &dataset.y, sizeof(int), 0,
                       hipMemcpyHostToDevice);

    // Allocate memory.

    // hipMalloc(&host_cells, sizeof(Cell *) * dataset.x * dataset.y);
    // hipMemcpyToSymbol(HIP_SYMBOL(cells), &host_cells, sizeof(Cell **), 0,
    //                    hipMemcpyHostToDevice);

    cells = (CellV **)my_obj_alloc.calloc<CellV *>(dataset.x * dataset.y);
    for (int i = 0; i < dataset.x * dataset.y; i++) {
        cells[i] = (Cell *)my_obj_alloc.my_new<Cell>();
        cells[i]->inst_cell(&my_obj_alloc);
        // assert(cells[i] != nullptr);
    }
    my_obj_alloc.toDevice();
    my_obj_alloc.create_tree();
    range_tree = my_obj_alloc.get_range_tree();
    tree_size = my_obj_alloc.get_tree_size();
    // Initialize cells.
    // create_cells<<<128, 128>>>();
    gpuErrchk(hipDeviceSynchronize());

    transfer_dataset();

    auto time_start = std::chrono::system_clock::now();
    printf("Checksum: %i\n", checksum());
    // Run simulation.
    for (int i = 0; i < kNumIterations; ++i) {
        candidate_prepare<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        alive_prepare<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        candidate_update<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        alive_update<<<1024, 1024>>>();
        gpuErrchk(hipDeviceSynchronize());

        // printf("Checksum: %i\n", checksum());
    }

    auto time_end = std::chrono::system_clock::now();
    auto elapsed = time_end - time_start;
    auto micros =
        std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

    printf("Checksum: %i\n", checksum());

    printf("%lu, \n", micros);

    return 0;
}
