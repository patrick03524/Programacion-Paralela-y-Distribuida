#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include "../../mem_alloc/mem_alloc.h"
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

class BaseClass {
  public:
    virtual __device__ void doTheMath(float &c, float a, int numCompute) = 0;
};

#define Derived(A)                                                             \
    class Class##A : public BaseClass {                                        \
      public:                                                                  \
        virtual __device__ void doTheMath(float &c, float a, int numCompute) { \
            for (int l = 0; l < numCompute; l++) c = c + a;                    \
        }                                                                      \
    }

Derived(0);
Derived(1);
Derived(2);
Derived(3);
Derived(4);
Derived(5);
Derived(6);
Derived(7);
Derived(8);
Derived(9);
Derived(10);
Derived(11);
Derived(12);
Derived(13);
Derived(14);
Derived(15);
Derived(16);
Derived(17);
Derived(18);
Derived(19);
Derived(20);
Derived(21);
Derived(22);
Derived(23);
Derived(24);
Derived(25);
Derived(26);
Derived(27);
Derived(28);
Derived(29);
Derived(30);
Derived(31);

#define ObjCase_cpu(A)                                         \
    case A:                                                    \
        if (numElements > i) {                                 \
            array[i] = (BaseClass *)alloc->my_new<Class##A>(); \
            break;                                             \
        }

#define ObjCase(A)                     \
    case A:                            \
        if (numElements > i) {         \
            new (array[i]) Class##A(); \
            break;                     \
        }

void initialize_0(BaseClass **pointerArray, int numElements, int numClasses,
                  int threadsPerBlock, obj_alloc *alloc) {
    int i;
    int threadIdx;
    BaseClass **array = pointerArray;
    for (i = 0; i < numElements; i++) {
        threadIdx = i / threadsPerBlock;
        switch (threadIdx % numClasses) {
            ObjCase_cpu(0);
            ObjCase_cpu(1);
            ObjCase_cpu(2);
            ObjCase_cpu(3);
            ObjCase_cpu(4);
            ObjCase_cpu(5);
            ObjCase_cpu(6);
            ObjCase_cpu(7);
            ObjCase_cpu(8);
            ObjCase_cpu(9);
            ObjCase_cpu(10);
            ObjCase_cpu(11);
            ObjCase_cpu(12);
            ObjCase_cpu(13);
            ObjCase_cpu(14);
            ObjCase_cpu(15);
            ObjCase_cpu(16);
            ObjCase_cpu(17);
            ObjCase_cpu(18);
            ObjCase_cpu(19);
            ObjCase_cpu(20);
            ObjCase_cpu(21);
            ObjCase_cpu(22);
            ObjCase_cpu(23);
            ObjCase_cpu(24);
            ObjCase_cpu(25);
            ObjCase_cpu(26);
            ObjCase_cpu(27);
            ObjCase_cpu(28);
            ObjCase_cpu(29);
            ObjCase_cpu(30);
            ObjCase_cpu(31);
        }
    }
}
__global__ void initialize_1(BaseClass **pointerArray, int numElements,
                             int numClasses) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    BaseClass **array = pointerArray;
    switch (threadIdx.x % numClasses) {
        ObjCase(0);
        ObjCase(1);
        ObjCase(2);
        ObjCase(3);
        ObjCase(4);
        ObjCase(5);
        ObjCase(6);
        ObjCase(7);
        ObjCase(8);
        ObjCase(9);
        ObjCase(10);
        ObjCase(11);
        ObjCase(12);
        ObjCase(13);
        ObjCase(14);
        ObjCase(15);
        ObjCase(16);
        ObjCase(17);
        ObjCase(18);
        ObjCase(19);
        ObjCase(20);
        ObjCase(21);
        ObjCase(22);
        ObjCase(23);
        ObjCase(24);
        ObjCase(25);
        ObjCase(26);
        ObjCase(27);
        ObjCase(28);
        ObjCase(29);
        ObjCase(30);
        ObjCase(31);
    }
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void ooVectorAdd(const float *A, float *C, int numElements,
                            BaseClass **classes, int numCompute) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    BaseClass *myClass = classes[i];
    if (i < numElements) {
        myClass->doTheMath(C[i], A[i], numCompute);
    }
}

/**
 * Host main routine
 */
int main(int argc, char **argv) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    mem_alloc shared_mem(4ULL * 1024 * 1024 * 1024);
    obj_alloc my_obj_alloc(&shared_mem);
    // Print the vector length to be used, and compute its size
    int numElements = atoi(argv[1]);  // size of vector
    int numCompute = atoi(argv[3]);   // vfunc body size
    int numClasses = atoi(argv[4]);   // num of types
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    hipDeviceSetLimit(hipLimitMallocHeapSize, 4ULL * 1024 * 1024 * 1024);
    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(
            stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(
            stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    BaseClass **classes = NULL;
    // hipMalloc((void***)&classes, sizeof(BaseClass*)*numElements);
    classes = (BaseClass **)my_obj_alloc.calloc<BaseClass *>(numElements);
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = atoi(argv[2]);  // thread per block
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    initialize_0(classes, numElements, numClasses, threadsPerBlock,
                 &my_obj_alloc);
    initialize_1<<<blocksPerGrid, threadsPerBlock>>>(classes, numElements,
                                                     numClasses);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch initialize kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
           threadsPerBlock);
    ooVectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements,
                                                    classes, numCompute);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to launch ooVectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(
            stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        float result = 0;
        for (int j = 0; j < numCompute; j++) result += h_A[i];
        if (fabs(result - h_C[i]) > 1e-3) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}
